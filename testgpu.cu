
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

#define N 65534
__global__ void add(int* a, int* b, int* c){
    int tid = blockIdx.x;
    if(tid<N)
        c[tid] = a[tid]*3 - b[tid]*2%2 + 111;
}

int main(void){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**) &dev_a, N*sizeof(int));
    hipMalloc((void**) &dev_b, N*sizeof(int));
    hipMalloc((void**) &dev_c, N*sizeof(int));

    for(int i=0; i<N ; i++){
        a[i] = i;
        b[i] = i;
    }
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    clock_t start, end;
    double duration;
    start = clock();
    add<<<N,1>>>(dev_a, dev_b, dev_c);

    end = clock();
    duration = (double)(end - start);
    printf("elapsed: %f", duration);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    //printf("vactor sum is:\n");
    //for(int i=0; i<N; i++){
    //    printf("%d  ", c[i]);

    //}
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
